#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define CHECK(call)                                                                       \
{                                                                                     \
    const hipError_t err = call;                                                     \
    if (err != hipSuccess)                                                           \
    {                                                                                 \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE);                                                           \
    }                                                                                 \
}
 
#define CHECK_KERNELCALL()                                                                \
{                                                                                     \
    const hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess)                                                           \
    {                                                                                 \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE);                                                           \
    }                                                                                 \
}

int main(int argc, const char *argv[])
{
    int finish = 0;

    int *d_finish;

    CHECK(hipMalloc(&d_finish, sizeof(int)));
    CHECK(hipMemset(d_finish,0, sizeof(int)));
    CHECK(hipMemcpy(&finish, d_finish, sizeof(int), hipMemcpyDeviceToHost));
    printf("Test finish dovrebbe essere = 1 invece è = %d\n", finish);

    return 0;
}