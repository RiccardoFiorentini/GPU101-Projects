
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <assert.h>

#define BLOCKN 8
#define THREADN 256

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }


double get_time() { // function to get the time of day in second
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, float **matrixDiagonal, const char *filename, int *num_rows, int *num_cols, int *num_vals){
    FILE *file = fopen(filename, "r");
    if (file == NULL){
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    // Get number of rows, columns, and non-zero values
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");

    //printf("Rows: %d, Columns:%d, NNZ:%d\n", *num_rows, *num_cols, *num_vals);
    int *row_ptr_t = (int *)malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *)malloc(*num_vals * sizeof(int));
    float *values_t = (float *)malloc(*num_vals * sizeof(float));
    float *matrixDiagonal_t = (float *)malloc(*num_rows * sizeof(float));
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *)malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++){
        row_occurances[i] = 0;
    }

    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF){
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        row_occurances[row]++;
    }

    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++){
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);

    // Set the file position to the beginning of the file
    rewind(file);

    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++){
        col_ind_t[i] = -1;
    }

    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int i = 0, j = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF){
        row--;
        column--;

        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1){
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        if (row == column){
            matrixDiagonal_t[j] = value;
            j++;
        }
        i = 0;
    }
    fclose(file);
    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;
    *matrixDiagonal = matrixDiagonal_t;
}

// CPU implementation of SYMGS using CSR, DO NOT CHANGE THIS
void symgs_csr_sw(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *matrixDiagonal){

    // forward sweep
    for (int i = 0; i < num_rows; i++){
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++){
            sum -= values[j] * x[col_ind[j]];
        }

        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }

    // backward sweep
    for (int i = num_rows - 1; i >= 0; i--){
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++){
            sum -= values[j] * x[col_ind[j]];
        }
        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }
}

__global__ void symgs_csr_gpu(const int *row_ptr, const int *col_ind, const float *values, const int *num_rows, float *x, float *matrixDiagonal, float* x2, char* locks, char* changed){
    int start, end, i;
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    int chunk_size = (int) *(num_rows) / (BLOCKN * THREADN);
    start = chunk_size * index;
    end = chunk_size * (index + 1);

    if(blockIdx.x == BLOCKN - 1 && threadIdx.x == THREADN - 1)
        end = *(num_rows);
    
    for(i = start; i < end; i++){
        locks[i] = 0;
        changed[i] = 0;
    }

    __syncthreads();
    
    char missed;
    do{
        missed = 0;
        for(i = start; i < end; i++){
            if(changed[i])
                continue;
                
            float sum = x[i];
            const int row_start = row_ptr[i];
            const int row_end = row_ptr[i + 1];
            float currentDiagonal = matrixDiagonal[i]; // Current diagonal value
    
            for (int j = row_start; j < row_end; j++){
                int index = col_ind[j];
                
                if(index < 0){
                    continue;
                }
                if(index < i){
                    if(locks[index] == 0){
                        missed = 1;
                        continue;
                    }
                    
                    sum -= (float) ((/*(double)*/ values[j]) * (/*(double)*/x2[index]));
                }
                else{
                    sum -= (float) ((/*(double)*/ values[j]) * (/*(double)*/x[index]));
                }
            }
            if(missed)
                continue;
            sum += (float) ((/*(double)*/ x[i]) * (/*(double)*/currentDiagonal));
            x2[i] = (float) ((/*(double)*/ sum) / (/*(double)*/currentDiagonal));
            locks[i] = 1;
            changed[i] = 1;
        }
    } while (missed);

    __syncthreads();

    do{
        missed = 0;
        for(i = end - 1; i >= start; i--){
            if(! changed[i])
                continue;
            
            float sum = x2[i];
            const int row_start = row_ptr[i];
            const int row_end = row_ptr[i + 1];
            float currentDiagonal = matrixDiagonal[i]; // Current diagonal value
    
            for (int j = row_start; j < row_end; j++){
                int index = col_ind[j];
                if(index < 0)
                    continue;
                if(index > i){
                    // new value is not ready yet, try next iteration
                    if(locks[index] == 1){
                        missed = 1;
                        continue;
                    }

                    sum -= (float)(/*(double)*/ values[j] * /*(double)*/ x[index]);
                }
                else
                    sum -= (float)(/*(double)*/ values[j] * /*(double)*/ x2[index]);
                
            }
            if(missed)
                continue;
            
            sum += (float) (/*(double)*/ x2[i] * /*(double)*/ currentDiagonal);
            x[i] = (float) (/*(double)*/ sum / /*(double)*/ currentDiagonal);
            locks[i] = 2;
            changed[i] = 0;
        }
    } while (missed);
}

int main(int argc, const char *argv[]){
    if (argc != 2){
        printf("Usage: ./exec matrix_file");
        return 0;
    }
    
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals;
    float *values;
    float *matrixDiagonal;
    
    const char *filename = argv[1];

    double start_cpu, end_cpu;
    double start_gpu, end_gpu;

    read_matrix(&row_ptr, &col_ind, &values, &matrixDiagonal, filename, &num_rows, &num_cols, &num_vals);
    float *x = (float *)malloc(num_rows * sizeof(float));
    float *xCopy = (float *)malloc(num_rows * sizeof(float));

    // Generate a random vector
    srand(time(NULL));
    for (int i = 0; i < num_rows; i++){
        x[i] = (float) (rand() % 100) / (float)(rand() % 100 + 1); // the number we use to divide cannot be 0, that's the reason of the +1
        xCopy[i] = x[i];
    }

    /* for(int i = 0; i < num_vals; i++){
        if(col_ind[i] < 0 || col_ind[i] > num_rows - 1){
            printf("%d, %d\n", i, col_ind[i]);
            assert(0);
            return 1;
        }
    }*/
    
    // Compute in sw
    start_cpu = get_time();
    symgs_csr_sw(row_ptr, col_ind, values, num_rows, x, matrixDiagonal);
    end_cpu = get_time();

    // gpu part
    
    // allocate space
    int *dev_row_ptr, *dev_col_ind, *dev_num_rows;
    float *dev_values, *dev_x, *dev_matrixDiagonal, *dev_x2;
    char *dev_locks, *dev_changed;
    CHECK(hipMalloc(&dev_row_ptr, (num_rows + 1) * sizeof(int)));
    CHECK(hipMalloc(&dev_col_ind, num_vals * sizeof(int)));
    CHECK(hipMalloc(&dev_values, num_vals * sizeof(float)));
    CHECK(hipMalloc(&dev_x, num_rows * sizeof(float)));
    CHECK(hipMalloc(&dev_matrixDiagonal, num_rows * sizeof(float)));
    CHECK(hipMalloc(&dev_x2, num_rows * sizeof(float)));
    CHECK(hipMalloc(&dev_locks, num_rows * sizeof(char)));
    CHECK(hipMalloc(&dev_changed, num_rows * sizeof(char)));
    CHECK(hipMalloc(&dev_num_rows, sizeof(int)));


    CHECK(hipMemcpy(dev_row_ptr, row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_col_ind, col_ind, num_vals * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_values, values, num_vals * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_x, xCopy, num_rows * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_matrixDiagonal, matrixDiagonal, num_rows * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_num_rows, &num_rows, sizeof(int), hipMemcpyHostToDevice));


    dim3 blocksPerGrid(BLOCKN, 1, 1);
    dim3 threadsPerBlock(THREADN, 1, 1);
    // compute in gpu
    start_gpu = get_time();
    
    symgs_csr_gpu<<<blocksPerGrid, threadsPerBlock>>>(
        dev_row_ptr,
        dev_col_ind,
        dev_values,
        dev_num_rows,
        dev_x,
        dev_matrixDiagonal,
        dev_x2,
        dev_locks,
        dev_changed
    );
    CHECK_KERNELCALL();


    end_gpu = get_time();
    
    CHECK(hipMemcpy(xCopy, dev_x, num_rows * sizeof(float), hipMemcpyDeviceToHost));
    
    FILE* output = fopen("./personal/errors.txt", "w");
    int errors = 0;
    float maxError = 0.0;
    for(int i = 0; i < num_rows; i++){
        if(x[i] - xCopy[i] > 0.0001 || x[i] - xCopy[i] < -0.0001 ){
            float err = x[i] - xCopy[i];
            err = err > 0 ? err : -err;
            maxError = err > maxError ? err : maxError;

            errors ++;
            if(errors < 100)
                fprintf(output, "WRONG RES ON GPU on x[i] for i = %d. x[i]=%.10lf, xCopy[i]=%.10lf\n", i, x[i], xCopy[i]); 
            //return 1;
        }
    }

    printf("Errors: %d\nMax error: %lf\n", errors, maxError);
    fclose(output);

    // Print time
    printf("SYMGS Time CPU: %.10lf\n", end_cpu - start_cpu);
    printf("SYMGS Time GPU: %.10lf\n", end_gpu - start_gpu);

    // Free: TODO
    free(row_ptr);
    free(col_ind);
    free(values);
    free(matrixDiagonal);

    CHECK(hipFree(dev_row_ptr));
    CHECK(hipFree(dev_col_ind));
    CHECK(hipFree(dev_values));
    CHECK(hipFree(dev_x));
    CHECK(hipFree(dev_matrixDiagonal));
    CHECK(hipFree(dev_x2));
    CHECK(hipFree(dev_locks));
    CHECK(hipFree(dev_changed));

    return 0;
}