#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, float **matrixDiagonal, const char *filename, int *num_rows, int *num_cols, int *num_vals)
{
    int err;
    FILE *file = fopen(filename, "r");
    if (file == NULL)
    {
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    // Get number of rows, columns, and non-zero values
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");

    int *row_ptr_t = (int *)malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *)malloc(*num_vals * sizeof(int));
    float *values_t = (float *)malloc(*num_vals * sizeof(float));
    float *matrixDiagonal_t = (float *)malloc(*num_rows * sizeof(float));
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *)malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++)
    {
        row_occurances[i] = 0;
    }

    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF)
    {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        row_occurances[row]++;
    }

    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++)
    {
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);

    // Set the file position to the beginning of the file
    rewind(file);

    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++)
    {
        col_ind_t[i] = -1;
    }

    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int i = 0, j = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF)
    {
        row--;
        column--;

        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1)
        {
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        if (row == column)
        {
            matrixDiagonal_t[j] = value;
            j++;
        }
        i = 0;
    }
    fclose(file);
    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;
    *matrixDiagonal = matrixDiagonal_t;
}

// CPU implementation of SYMGS using CSR, DO NOT CHANGE THIS
void symgs_csr_sw(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *matrixDiagonal)
{

    // forward sweep
    for (int i = 0; i < num_rows; i++)
    {
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++)
        {
            sum -= values[j] * x[col_ind[j]];
        }

        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }

    // backward sweep
    for (int i = num_rows - 1; i >= 0; i--)
    {
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++)
        {
            sum -= values[j] * x[col_ind[j]];
        }
        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }
}


//implementation of the first part of the algorithm
__device__ void forwardSweep(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *y, float *matrixDiagonal)
{
    const int row = blockIdx.x*blockDim.x + threadIdx.x;
    if(row<num_rows){    
        float tmp = x[row];
        const int row_start = row_ptr[row];
        const int row_end = row_ptr[row+1];

        for(int col = row_start; col < row_end; col++){
            tmp -= values[col]*x[col_ind[col]];
        }

        tmp = (tmp + (x[row] * matrixDiagonal[row])) / matrixDiagonal[row];
        y[row] = tmp;
    }
}

//da pensare come ottimizzare
__device__ void backwardSweep(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *matrixDiagonal)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    float tmp;
    int row_start = row_ptr[row];
    int row_end = row_ptr[row+1];
    if(row<num_rows){
        tmp = internalFor(row_start - row_end, col_ind + row_start, values + row_start, y, row);
        tmp = (tmp + (y[row] * matrixDiagonal[row])) / matrixDiagonal[row];
    }
    
    if(row<num_rows){
        y[row] = tmp;
    }
}


// GPU implementation of SYMGS using CSR
/**
 * @brief 
 * 
 * @param row_ptr pointer to row starts in col_ind and data + last item = to muber of values
 * @param col_ind array of column indeces
 * @param values array of corresponding nonzero values
 * @param num_rows number of rows
 * @param x vector to multiply for
 * @param matrixDiagonal vector of diagonal values
 * @return __global__ 
 */
__global__ void symgsGPU(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *y, float *matrixDiagonal)
{
    //forwardSweep
    forwardSweep(row_ptr, col_ind, values, num_rows, x, y, matrixDiagonal);
}



int main(int argc, const char *argv[])
{

    if (argc != 2)
    {
        printf("Usage: ./exec matrix_file");
        return 0;
    }

    //CPU vaiables
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals;
    float *values;
    float *matrixDiagonal;

    const char *filename = argv[1];

    double start_time, end_time;

    //lettuta matrice e inizializzazione variabili
    read_matrix(&row_ptr, &col_ind, &values, &matrixDiagonal, filename, &num_rows, &num_cols, &num_vals);
    float *x = (float *)malloc(num_rows * sizeof(float));

    //Generate a random vector
    srand(time(NULL));
    for (int i = 0; i < num_rows; i++)
    {
        x[i] = (rand() % 100) / (rand() % 100 + 1); // the number we use to divide cannot be 0, that's the reason of the +1
    }

    //GPU vaiables
    int *d_row_ptr, *d_col_ind;
    float *d_values;
    float *d_matrixDiagonal;
    float *d_x, *d_y;

    //allocazione memoria per vettori su gpu
    hipMalloc(&d_row_ptr, (num_rows + 1) * sizeof(int));
    hipMalloc(&d_col_ind, num_vals * sizeof(int));
    hipMalloc(&d_values, num_vals * sizeof(float));
    hipMalloc(&d_matrixDiagonal, num_rows * sizeof(float));
    hipMalloc(&d_x, num_rows * sizeof(float));
    hipMalloc(&d_y, num_rows * sizeof(float));

    //copia e inizializzazione vettori su gpu
    hipMemcpy(d_row_ptr, row_ptr,  (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, col_ind,  num_vals * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, values,  num_vals * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrixDiagonal, matrixDiagonal,  num_rows * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x,  num_rows * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, num_rows * sizeof(float));

    start_time = get_time();
    dim3 blocksPerGrid(num_rows/1024, 1, 1);
    dim3 ThreadsPerBlock(1024, 1, 1);
    symgsGPU<<<blocksPerGrid, ThreadsPerBlock>>>(d_row_ptr, d_col_ind, d_values, num_rows, d_x, d_y, d_matrixDiagonal);
    end_time = get_time();
    printf("SYMGS Time GPU: %.10lf\n", end_time - start_time);

    //creo vettore di supporto per il testing e copio il risultato della gpu
    float *y = (float *)malloc(num_rows * sizeof(float));
    hipMemcpy(y, d_y,  num_rows * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_values);
    hipFree(d_matrixDiagonal);
    hipFree(d_x);
    hipFree(d_y);

    // Compute in sw
    start_time = get_time();
    symgs_csr_sw(row_ptr, col_ind, values, num_rows, x, matrixDiagonal);
    end_time = get_time();

    // Print time
    printf("SYMGS Time CPU: %.10lf\n", end_time - start_time);

    bool correct = true;
    int i = 0;
    for(i = 0; i<num_rows && correct; i++){
        correct = (y[i] == x[i]);
    }

    if(correct){
        printf("Yeeeee\n");
    }else{
        printf("fail\n");
        printf("error at %d. Val x = %f, Val y = %f \n", i-1, x[i-1], y[i-1]);
    }
   
    free(row_ptr);
    free(col_ind);
    free(values);
    free(matrixDiagonal);
    free(y);
    free(x);

    return 0;
}