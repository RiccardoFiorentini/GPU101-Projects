#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

const int RPT = 64;
const int NUMTHR = 512;

#define CHECK(call)                                                                       \
{                                                                                     \
    const hipError_t err = call;                                                     \
    if (err != hipSuccess)                                                           \
    {                                                                                 \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE);                                                           \
    }                                                                                 \
}

#define CHECK_KERNELCALL()                                                                \
{                                                                                     \
    const hipError_t err = hipGetLastError();                                       \
    if (err != hipSuccess)                                                           \
    {                                                                                 \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        exit(EXIT_FAILURE);                                                           \
    }                                                                                 \
}

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}
 
// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, float **matrixDiagonal, const char *filename, int *num_rows, int *num_cols, int *num_vals)
{
    FILE *file = fopen(filename, "r");
    if (file == NULL)
    {
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    // Get number of rows, columns, and non-zero values
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");

    int *row_ptr_t = (int *)malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *)malloc(*num_vals * sizeof(int));
    float *values_t = (float *)malloc(*num_vals * sizeof(float));
    float *matrixDiagonal_t = (float *)malloc(*num_rows * sizeof(float));
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *)malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++)
    {
        row_occurances[i] = 0;
    }

    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF)
    {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        row_occurances[row]++;
    }

    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++)
    {
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);

    // Set the file position to the beginning of the file
    rewind(file);

    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++)
    {
        col_ind_t[i] = -1;
    }

    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int i = 0, j = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF)
    {
        row--;
        column--;

        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1)
        {
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        if (row == column)
        {
            matrixDiagonal_t[j] = value;
            j++;
        }
        i = 0;
    }
    fclose(file);
    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;
    *matrixDiagonal = matrixDiagonal_t;
}

// CPU implementation of SYMGS using CSR, DO NOT CHANGE THIS
void symgs_csr_sw(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *matrixDiagonal)
{

    // forward sweep
    for (int i = 0; i < num_rows; i++)
    {
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++)
        {
            sum -= values[j] * x[col_ind[j]];
        }

        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }

    // backward sweep
    for (int i = num_rows - 1; i >= 0; i--)
    {
        float sum = x[i];
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        float currentDiagonal = matrixDiagonal[i]; // Current diagonal value

        for (int j = row_start; j < row_end; j++)
        {
            sum -= values[j] * x[col_ind[j]];
        }
        sum += x[i] * currentDiagonal; // Remove diagonal contribution from previous loop

        x[i] = sum / currentDiagonal;
    }
}


//implementation of the first part of the algorithm
__global__ void forwardSweep(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, bool *modified, float *matrixDiagonal)
{
    const int row1 = (blockIdx.x*blockDim.x + threadIdx.x)*RPT;
    const int rowLast = row1 + RPT;

    if(row1>=0 && row1<num_rows){
        for(int row = row1; row < rowLast && row<num_rows; row++){
            printf("Riga eseguita: %d\n", row);

            float tmp = x[row];
            const int row_start = row_ptr[row];
            const int row_end = row_ptr[row+1];
            bool process = true;

            for(int col = row_start; col < row_end; col++){
                while(process){
                    if(modified[col_ind[col]]==true || col_ind[col]<=row){
                        tmp -= values[col]*x[col_ind[col]];
                        process = false;
                    }
                }
                process = true;
            }

            process = true;
            while(process){
                if(row == (num_rows-1) || modified[row+1]==true){
                    tmp += x[row] * matrixDiagonal[row];
                    x[row] = tmp / matrixDiagonal[row];
                    modified[row] = true;
                    process = false;
                }
            }
        }
    }
    printf("fine thread #%d\n", (blockIdx.x*blockDim.x + threadIdx.x));
}

//da pensare come ottimizzare
__global__ void backwardSweep(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, bool *modified, float *matrixDiagonal)
{
    const int row1 = num_rows - 1 - (blockIdx.x*blockDim.x + threadIdx.x)*RPT;
    const int rowLast = row1 + RPT;
    if(row1>=0 && row1<num_rows){ //non consideri se rimane a metà
        int row = 0;
        for(row = rowLast; row>=row1; row--){
            float tmp = x[row];
            const int row_start = row_ptr[row];
            const int row_end = row_ptr[row+1];
            bool process;

            for(int col = row_start; col < row_end; col++){
                process = true;
                while(process){
                    if(modified[col_ind[col]]==true || col_ind[col]<=row){
                        tmp -= values[col]*x[col_ind[col]];
                        process = false;
                    }
                }
            }

            process = true;
            while(process){
                if(row == 0 || modified[row+1]==true){
                    tmp += x[row] * matrixDiagonal[row];
                    x[row] = tmp / matrixDiagonal[row];
                    modified[row] = true;
                    process = false;
                }
            }
        }
    }
    printf("fine thread #%d\n", (blockIdx.x*blockDim.x + threadIdx.x));
}


// GPU implementation of SYMGS using CSR
/**
 * @brief 
 * 
 * @param row_ptr pointer to row starts in col_ind and data + last item = to muber of values
 * @param col_ind array of column indeces
 * @param values array of corresponding nonzero values
 * @param num_rows number of rows
 * @param x vector to multiply for
 * @param matrixDiagonal vector of diagonal values
 * @return __global__ 
 */
/*
__global__ void symgsGPU(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, float *x, float *y, float *matrixDiagonal)
{
    //forwardSweep
    forwardSweep(row_ptr, col_ind, values, num_rows, x, y, matrixDiagonal);
}*/


int main(int argc, const char *argv[])
{
    if (argc != 2)
    {
        printf("Usage: ./exec matrix_file");
        return 0;
    }

    //CPU vaiables
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals;
    float *values;
    float *matrixDiagonal;

    const char *filename = argv[1];

    double start_time, end_time;

    //lettuta matrice e inizializzazione variabili
    read_matrix(&row_ptr, &col_ind, &values, &matrixDiagonal, filename, &num_rows, &num_cols, &num_vals);
    float *x = (float *)malloc(num_rows * sizeof(float));
    printf("END reading matrix\n");
    //Generate a random vector
    srand(time(NULL));
    for (int i = 0; i < num_rows; i++)
    {
        x[i] = (rand() % 100) / (rand() % 100 + 1); // the number we use to divide cannot be 0, that's the reason of the +1
    }
    
    printf("X generated\n");

    //GPU vaiables
    int *d_row_ptr, *d_col_ind;
    float *d_values;
    float *d_matrixDiagonal;
    float *d_x;
    bool  *d_modified;
 
    //allocazione memoria per vettori su gpu
    CHECK(hipMalloc(&d_row_ptr, (num_rows + 1) * sizeof(int)));
    CHECK(hipMalloc(&d_col_ind, num_vals * sizeof(int)));
    CHECK(hipMalloc(&d_values, num_vals * sizeof(float)));
    CHECK(hipMalloc(&d_matrixDiagonal, num_rows * sizeof(float)));
    CHECK(hipMalloc(&d_x, num_rows * sizeof(float)));
    CHECK(hipMalloc(&d_modified, num_rows * sizeof(bool)));

    //copia e inizializzazione vettori su gpu
    CHECK(hipMemcpy(d_row_ptr, row_ptr,  (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_col_ind, col_ind,  num_vals * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_values, values,  num_vals * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_matrixDiagonal, matrixDiagonal,  num_rows * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_x, x,  num_rows * sizeof(float), hipMemcpyHostToDevice));
//    CHECK(cudaMemset(d_modified, 0, num_rows * sizeof(double)));

    printf("GPU START\n");
    start_time = get_time();
    dim3 blocksPerGrid(num_rows/(NUMTHR*RPT), 1, 1);
    dim3 ThreadsPerBlock(NUMTHR, 1, 1);
    forwardSweep<<<blocksPerGrid, ThreadsPerBlock>>>(d_row_ptr, d_col_ind, d_values, num_rows, d_x, d_modified, d_matrixDiagonal);
    CHECK_KERNELCALL();
    hipDeviceSynchronize();
    backwardSweep<<<blocksPerGrid, ThreadsPerBlock>>>(d_row_ptr, d_col_ind, d_values, num_rows, d_x, d_modified, d_matrixDiagonal);
    CHECK_KERNELCALL();
    hipDeviceSynchronize();
    end_time = get_time();
    printf("SYMGS Time GPU: %.10lf\n", end_time - start_time);

    //creo vettore di supporto per il testing e copio il risultato della gpu
    float *y = (float *)malloc(num_rows * sizeof(float));
    hipMemcpy(y, d_x,  num_rows * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_values);
    hipFree(d_matrixDiagonal);
    hipFree(d_x);
    hipFree(d_modified);

    // Compute in sw
    start_time = get_time();
    symgs_csr_sw(row_ptr, col_ind, values, num_rows, x, matrixDiagonal);
    end_time = get_time();

    // Print time
    printf("SYMGS Time CPU: %.10lf\n", end_time - start_time);

    bool correct = true;
    int i = 0;
    for(i = 0; i<num_rows && correct; i++){
        correct = (y[i] == x[i]);
    }

    if(correct){
        printf("Yeeeee\n");
    }else{
        printf("fail\n");
        printf("error at %d. Val x = %f, Val y = %f \n", i-1, x[i-1], y[i-1]);
    }
   
    free(row_ptr);
    free(col_ind);
    free(values);
    free(matrixDiagonal);
    free(y);
    free(x);

    return 0;
}